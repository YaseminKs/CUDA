#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>

#define N 8  // Must be a power of 2

__global__ void fftKernel( hipComplex *X, int n, int step ){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if( tid < n / 2 ){
        int pos = tid * step * 2;
        hipComplex even = X[pos];
        hipComplex odd = X[pos + step];

        float angle = -2.0f * M_PI * tid / n;
        hipComplex twiddle = make_hipComplex( cosf( angle ), sinf( angle ) );
        hipComplex temp = hipCmulf( twiddle, odd );

        X[pos] = hipCaddf( even, temp );
        X[pos + step] = hipCsubf( even, temp );
    }
}

void cudaFFT( hipComplex *h_X ){
    hipComplex *d_X;
    hipMalloc( ( void** )&d_X, N * sizeof( hipComplex ) );
    hipMemcpy( d_X, h_X, N * sizeof( hipComplex ), hipMemcpyHostToDevice );

    for( int step = 1 ; step < N ; step *= 2 ){
        fftKernel<<<1, N / 2>>>( d_X, N, step );
        hipDeviceSynchronize();
    }

    hipMemcpy( h_X, d_X, N * sizeof( hipComplex ), hipMemcpyDeviceToHost );
    hipFree( d_X );
}

int main(){
    hipComplex h_X[N];
    for( int i = 0 ; i < N ; i++ ){
        h_X[i] = make_hipComplex( i, 0 ); // Example: Real input, imaginary part = 0
    }

    printf( "Input:\n" );
    for( int i = 0 ; i < N ; i++ ){
        printf( "(%f, %f) ", hipCrealf( h_X[i] ), hipCimagf( h_X[i] ) );
    }
    printf( "\n" );

    cudaFFT( h_X );

    printf( "FFT Output:\n" );
    for( int i = 0 ; i < N ; i++ ){
        printf( "(%f, %f) ", hipCrealf( h_X[i] ), hipCimagf( h_X[i] ) );
    }
    printf( "\n" );

    return 0;
}
