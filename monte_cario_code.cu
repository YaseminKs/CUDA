#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define N 1000000

__global__ void monteCarloKernel( int *d_count, int n ){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState state;
    hiprand_init( 1234, tid, 0, &state );

    if( tid < n ){
        float x = hiprand_uniform( &state );
        float y = hiprand_uniform( &state );
        if( x * x + y * y <= 1.0f )
            atomicAdd( d_count, 1 );
    }
}

void cudaMonteCarlo(){
    int h_count = 0, *d_count;
    hipMalloc( ( void** )&d_count, sizeof( int ) );
    hipMemcpy( d_count, &h_count, sizeof( int ), hipMemcpyHostToDevice );

    monteCarloKernel<<<( N + 255 ) / 256, 256>>>( d_count, N );
    hipMemcpy( &h_count, d_count, sizeof( int ), hipMemcpyDeviceToHost );

    hipFree( d_count );
    printf( "Estimated Pi: %f\n", ( 4.0f * h_count ) / N );
}

int main(){
    cudaMonteCarlo();
    return 0;
}
